#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand_kernel.h>

#include "voxel_upsample_gpu.h"

#define THREADS_PER_BLOCK 256
#define DIVUP(m,n) ((m) / (n) + ((m) % (n) > 0))

__global__ void voxel_upsampler_kernel(int M, int R1, int R2, int R3, int nsample, 
            float radius, int z_range, int y_range, int x_range, const float *new_xyz, 
            const float *xyz, const int *new_coords, const int *point_indices, 
            const float *voxel_score, float *pt_score, float *distance) {
    // :param new_coords: (M1 + M2 ..., 4) centers of the ball query
    // :param point_indices: (B, Z, Y, X)
    // output:
    //      idx: (M1 + M2, nsample)
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (pt_idx >= M) return;
    
    new_xyz += pt_idx * 3;
    new_coords += pt_idx * 3;
    pt_score += pt_idx * nsample;
    distance += pt_idx * nsample;

    float radius2 = radius * radius;
    float new_x = new_xyz[0];
    float new_y = new_xyz[1];
    float new_z = new_xyz[2];

    int new_coords_z = new_coords[0];
    int new_coords_y = new_coords[1];
    int new_coords_x = new_coords[2];
    
    int cnt = 0;
    // for (int dz = -1*z_range; dz <= z_range; ++dz) {
    for (int dz = -1*z_range; dz <= z_range; ++dz) {
        int z_coord = new_coords_z + dz;
        if (z_coord < 0 || z_coord >= R1) continue;

        for (int dy = -1*y_range; dy <= y_range; ++dy) {
            int y_coord = new_coords_y + dy;
            if (y_coord < 0 || y_coord >= R2) continue;

            for (int dx = -1*x_range; dx <= x_range; ++dx) {
                int x_coord = new_coords_x + dx;
                if (x_coord < 0 || x_coord >= R3) continue;

                int index = z_coord * R2 * R3 + \
                            y_coord * R3 + \
                            x_coord;
                int neighbor_idx = point_indices[index];
                if (neighbor_idx < 0) continue;
                
                float x_per = xyz[neighbor_idx*3 + 0];
                float y_per = xyz[neighbor_idx*3 + 1];
                float z_per = xyz[neighbor_idx*3 + 2];

                float dist2 = (x_per - new_x) * (x_per - new_x) + (y_per - new_y) * (y_per - new_y) + (z_per - new_z) * (z_per - new_z);

                //#if (dist2 > radius2) continue;
                
                if (cnt < nsample) {
                    pt_score[cnt] = voxel_score[neighbor_idx];
                    distance[cnt] = dist2;
                    ++cnt;
                }
            }
        }
    }
}


void voxel_upsampler_kernel_launcher(int M, int R1, int R2, int R3, int nsample,
    float radius, int z_range, int y_range, int x_range, const float *new_xyz, 
    const float *xyz, const int *new_coords, const int *point_indices, 
    const float *voxel_score, float *pt_score, float *distance){
    // :param new_coords: (M1 + M2 ..., 4) centers of the voxel query
    // :param point_indices: (B, Z, Y, X) 
    // output:
    //      pt_score: (M1 + M2, nsample)
    //      distance: (M1 + M2, nsample)

    hipError_t err;

    dim3 blocks(DIVUP(M, THREADS_PER_BLOCK));  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    voxel_upsampler_kernel<<<blocks, threads>>>(M, R1, R2, R3, nsample, radius, z_range, y_range, 
                                                x_range, new_xyz, xyz, new_coords, point_indices, 
                                                voxel_score, pt_score, distance);
    hipDeviceSynchronize();  // for using printf in kernel function

    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}
